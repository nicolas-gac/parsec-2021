#include "mylib.h"
#include "mylib.cuh"


Mat seuillageGPU( Mat in)
{
	hipError_t error;
	Mat out;
	out.create(in.rows,in.cols,CV_8UC3);
	
	// allocate host memory
	unsigned char *h_image_in_GPU ;
	h_image_in_GPU=in.data;
	
	/*hipEvent_t start,stop,start_mem,stop_mem;
	error = hipEventCreate(&start_mem);
	error = hipEventCreate(&stop_mem);
	
	error = hipEventRecord(start, NULL);
	error = hipEventSynchronize(start);*/
	
	// images on device memoryÍÍÍ
	unsigned char *d_image_in_GPU;
	unsigned char *d_image_out_GPU;
	
	const unsigned long int mem_size=in.cols*in.rows*3*sizeof(unsigned char);
	
	// Alocation mémoire de d_image_in et d_image_out sur la carte GPU
	hipMalloc((void**) &d_image_in_GPU,mem_size );
	hipMalloc((void**) &d_image_out_GPU, mem_size);
	
	// copy host memory to device
	hipMemcpy(d_image_in_GPU, h_image_in_GPU,mem_size ,hipMemcpyHostToDevice);
	
	//error = hipEventRecord(stop_mem, NULL);
	
	// Wait for the stop event to complete
	//error = hipEventSynchronize(stop_mem);
	//float msecMem = 0.0f;
	//error = hipEventElapsedTime(&msecMem, start, stop_mem);
	
	// setup execution parameters -> découpage en threads
	dim3 threads(BLOCK_SIZE,BLOCK_SIZE);
	dim3 grid(in.rows/BLOCK_SIZE,in.cols/BLOCK_SIZE);
	
	// lancement des threads executé sur la carte GPU
	kernel_seuillageGPU<<< grid, threads >>>(d_image_in_GPU, d_image_out_GPU,in.cols);
	
	// Record the start event
	//error = hipEventRecord(start_mem, NULL);
	//error = hipEventSynchronize(start_mem);
	
	// copy result from device to host
	hipMemcpy(out.data, d_image_out_GPU, mem_size,hipMemcpyDeviceToHost);
	hipFree(d_image_in_GPU);
	hipFree(d_image_out_GPU);
	/*
	float msecTotal,msecMem2;
	error = hipEventRecord(stop, NULL);
	error = hipEventSynchronize(stop);
	error = hipEventElapsedTime(&msecTotal, start, stop);
	error = hipEventElapsedTime(&msecMem2, start_mem, stop);
	*/
	return out;
}

