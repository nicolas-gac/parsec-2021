#include "hip/hip_runtime.h"
#include "mylib.h"
#include "mylib.cuh"
//---------------------noirBlanc-----------------------

Mat noirBlanc(Mat frame)
{ 
	Mat im_gray_out;
	
	if (frame.empty())
	exit(0);
	
	cvtColor(frame,im_gray_out,CV_RGB2GRAY);	
	return im_gray_out;
}



//---------------------get_frame-----------------------

Mat get_frame(Mat frame)
{
	if (frame.empty())
	exit(0);
	
	return frame;
}

//---------------------seuillage------------------------

Mat seuillage(Mat frame)
{ 
	float nr;
	uchar r,v,b;
	
	if (frame.empty())
	exit(0);
	
	for(int i = 0; i < frame.rows; i++)
	{
		for(int j = 0; j < frame.cols; j++)
		{
			
			b = frame.at<Vec3b>(i,j)[0];
			v = frame.at<Vec3b>(i,j)[1];
			r = frame.at<Vec3b>(i,j)[2];
			
			nr = v/sqrt(r*r+b*b+v*v);
			
			if (nr > 0.6){
				frame.at<Vec3b>(i,j)[0] = b;
				frame.at<Vec3b>(i,j)[1] = r;
				frame.at<Vec3b>(i,j)[2] = v;
			}
			else{
				frame.at<Vec3b>(i,j)[0] = b;
				frame.at<Vec3b>(i,j)[1] = v;
				frame.at<Vec3b>(i,j)[2] = r;
			}
		}
	}
	return frame;
}


//---------------------contour------------------------

Mat contour(Mat frame)
{ 
	
	Mat frame_out,frame_grayt;
	
	cvtColor(frame,frame_grayt,CV_BGR2GRAY);
	frame_out.create(frame.rows,frame.cols,CV_8UC1);
	
	if (frame.empty())
	exit(0);
	
	for (int i=1;i<frame.rows;i++){
		for (int j=1;j<frame.cols;j++){
			short temp;
			temp = (-1)*(short)frame_grayt.at<uchar>(i,j-1)+(-1)*(short)frame_grayt.at<uchar>(i-1,j)+(-1)*(char)frame_grayt.at<uchar>(i,j+1)+(-1)*(short)frame_grayt.at<uchar>(i+1,j)+4*(short)frame_grayt.at<uchar>(i,j);
			
			frame_out.at<uchar>(i,j)=(uchar)abs(temp);
			
			if(frame_out.at<uchar>(i,j)>23) frame_out.at<uchar>(i,j)=255;
			else frame_out.at<uchar>(i,j)=0;
		}
	}
	return frame_out;
}


Mat seuillageGPU( Mat in)
{
	hipError_t error;
	Mat out;
	out.create(in.rows,in.cols,CV_8UC3);
	
	// allocate host memory
	unsigned char *h_image_in_GPU ;
	h_image_in_GPU=in.data;
	
	hipEvent_t start,stop,start_mem,stop_mem;
	error = hipEventCreate(&start_mem);
	error = hipEventCreate(&stop_mem);
	
	error = hipEventRecord(start, NULL);
	error = hipEventSynchronize(start);
	
	// images on device memoryÍÍÍ
	unsigned char *d_image_in_GPU;
	unsigned char *d_image_out_GPU;
	
	const unsigned long int mem_size=in.cols*in.rows*3*sizeof(unsigned char);
	
	// Alocation mémoire de d_image_in et d_image_out sur la carte GPU
	hipMalloc((void**) &d_image_in_GPU,mem_size );
	hipMalloc((void**) &d_image_out_GPU, mem_size);
	
	// copy host memory to device
	hipMemcpy(d_image_in_GPU, h_image_in_GPU,mem_size ,hipMemcpyHostToDevice);
	
	error = hipEventRecord(stop_mem, NULL);
	
	// Wait for the stop event to complete
	error = hipEventSynchronize(stop_mem);
	float msecMem = 0.0f;
	error = hipEventElapsedTime(&msecMem, start, stop_mem);
	
	// setup execution parameters -> découpage en threads
	dim3 threads(BLOCK_SIZE,BLOCK_SIZE);
	dim3 grid(in.rows/BLOCK_SIZE,in.cols/BLOCK_SIZE);
	
	// lancement des threads executé sur la carte GPU
	kernel_seuillageGPU<<< grid, threads >>>(d_image_in_GPU, d_image_out_GPU,in.cols);
	
	// Record the start event
	error = hipEventRecord(start_mem, NULL);
	error = hipEventSynchronize(start_mem);
	
	// copy result from device to host
	hipMemcpy(out.data, d_image_out_GPU, mem_size,hipMemcpyDeviceToHost);
	hipFree(d_image_in_GPU);
	hipFree(d_image_out_GPU);
	float msecTotal,msecMem2;
	error = hipEventRecord(stop, NULL);
	error = hipEventSynchronize(stop);
	error = hipEventElapsedTime(&msecTotal, start, stop);
	error = hipEventElapsedTime(&msecMem2, start_mem, stop);
	
	return out;
}

