#include "mylib.h"
#include <hip/hip_runtime.h>

// acces au flux de la camera
std::string gstreamer_pipeline (int capture_width, int capture_height, int display_width, int display_height, int framerate, int flip_method) {
    return "nvarguscamerasrc ! video/x-raw(memory:NVMM), width=(int)" + std::to_string(capture_width) + ", height=(int)" +
           std::to_string(capture_height) + ", format=(string)NV12, framerate=(fraction)" + std::to_string(framerate) +
           "/1 ! nvvidconv flip-method=" + std::to_string(flip_method) + " ! video/x-raw, width=(int)" + std::to_string(display_width) + ", height=(int)" +
           std::to_string(display_height) + ", format=(string)BGRx ! videoconvert ! video/x-raw, format=(string)BGR ! appsink";
}

int main(int, char**)
{ 
    	int capture_width = 1280 ;
    	int capture_height = 720 ;
    	int display_width = 640 ;
    	int display_height = 360 ;
    	int framerate = 60 ;
    	int flip_method = 0 ;

    	std::string pipeline = gstreamer_pipeline(capture_width,
	capture_height,
	display_width,
	display_height,
	framerate,
	flip_method);
    	std::cout << "Using pipeline: \n\t" << pipeline << "\n";

	cv::VideoCapture cap(pipeline, cv::CAP_GSTREAMER);
	//VideoCapture cap(); // open the default camera
	if(!cap.isOpened())  // check if we succeeded
		return -1;
	
	while(1){
		Mat frame;
	    	cap >> frame;
	    	char c=(char)waitKey(25);
		if(c == 97){				// if 'a' est appuye
			Mat NB = noirBlanc(frame);
	    		imshow("NoirEtBlanc", NB);
		}
		else if(c == 98){			// if 'b' est appuye
			Mat seuil = seuillage(frame);
	    		imshow("seuillage", seuil);
		}
		else if (c == 99){			// if 'c' est appuye
			Mat cont = contour(frame);
	    		imshow("contour", cont);
		}
		/*else if (c == 100){			// if 'd' est appuye
			Mat seuilgpu = seuillageGPU(frame);
	    		imshow("seuillage GPU",seuilgpu);*/
;
		}
		else if(c == 101) destroyAllWindows();	// if 'e' est appuye

		else imshow("frame", frame);
			
	    	if(c==27)				// if 'esc' est appuye
	      		break;
	}
  	// When everything done, release the video capture object
	cap.release();

	// Closes all the frames
	destroyAllWindows();

	return 0;
}
